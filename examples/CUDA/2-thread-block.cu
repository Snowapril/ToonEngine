
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUM_BLOCKS 16
#define BLOCK_WIDTH 1

__global__
void hello() 
{
    printf("Hello, i am a thread in a block %d\n", blockIdx.x);
}

int main(int argc, char** argv)
{
    hello<<< NUM_BLOCKS, BLOCK_WIDTH >>>();
    hipDeviceSynchronize();
    printf("That's all!");
    return 0;
}