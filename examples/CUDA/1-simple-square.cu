
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void square(float *d_in, float *d_out)
{
    int idx = threadIdx.x;
    float v = d_in[idx];
    d_out[idx] = v * v;
}

int main(int argc, char **argv)
{
    const int ARRAY_SIZE  = 64;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

    // generate the input array on the host
    float h_in[ARRAY_SIZE], h_out[ARRAY_SIZE];
    for (int i = 0; i < ARRAY_SIZE; i++)
        h_in[i] = i + 1;
    
    // declare GPU memory pointers.
    float *d_in, *d_out;
    // allocate GPU memory
    hipMalloc((void**)&d_in, ARRAY_BYTES);
    hipMalloc((void**)&d_out, ARRAY_BYTES);

    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);
    square<<< 1, ARRAY_SIZE >>>(d_in, d_out);
    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

    for (int i = 0; i < ARRAY_SIZE; i++)
        printf("%d th array element : %f\n", i, h_out[i]);

    hipDeviceSynchronize();

    return 0;
}