
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUM_THREADS 1000000
#define ARRAY_SIZE  10
#define BLOCK_WIDTH 1000

void print_array(int *arr, int size)
{
    int i;
    for (i = 0; i < size; i++)
    {
        printf("array[%d] = %d\n", i, arr[i]);
    }
}

__global__ void increment_naive(int* g)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    i = i % ARRAY_SIZE;
    g[i] = g[i] + 1;
}

__global__ void increment_sync(int* g)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    i = i % ARRAY_SIZE;
    int temp = g[i];
    __syncthreads();
    g[i] = temp + 1;
    __syncthreads();
}

__global__ void increment_atomic(int* g)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    i = i % ARRAY_SIZE;
    atomicAdd(&g[i], 1);
}

int main(int argc, char** argv)
{
    printf("%d total threads in %d blocks writing into %d array elements\n", 
            NUM_THREADS, NUM_THREADS / BLOCK_WIDTH, ARRAY_SIZE);
    
    int h_array[ARRAY_SIZE];
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);

    int *d_array;
    hipMalloc((void**)&d_array, ARRAY_BYTES);
    hipMemset((void*)d_array, 0, ARRAY_BYTES);

    const int BLOCK_DIM = NUM_THREADS / BLOCK_WIDTH;

    printf("\nwithout __syncthreads or atomic operation\n");
    increment_naive<<<BLOCK_DIM, BLOCK_WIDTH>>>(d_array);
    hipMemcpy(h_array, d_array, ARRAY_BYTES, hipMemcpyDeviceToHost);
    print_array(h_array, ARRAY_SIZE);

    printf("\nwith __syncthreads\n");
    increment_sync<<<BLOCK_DIM, BLOCK_WIDTH>>>(d_array);
    hipMemcpy(h_array, d_array, ARRAY_BYTES, hipMemcpyDeviceToHost);
    print_array(h_array, ARRAY_SIZE);

    printf("\nwith atomic operations\n");
    increment_atomic<<<BLOCK_DIM, BLOCK_WIDTH>>>(d_array);
    hipMemcpy(h_array, d_array, ARRAY_BYTES, hipMemcpyDeviceToHost);
    print_array(h_array, ARRAY_SIZE);

    hipFree(d_array);
    return 0;
}